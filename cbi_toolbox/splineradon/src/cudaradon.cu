#include "hip/hip_runtime.h"
//
// Created by fmarelli on 05/07/19.
//
#include "cudaradon.h"

#include <hip/hip_runtime.h>
#include <>

#include <math.h>


// Throw std::runtime_error when a cuda error occurs
#define CUDA_EXCEPTIONS

#define MAX(A, B)  ((A) > (B) ? (A) : (B))
#define MIN(A, B)  ((A) < (B) ? (A) : (B))


inline
hipError_t checkCuda(hipError_t result) {
#if defined(CUDA_EXCEPTIONS)
    if (result != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(result));
    }
#endif
    return result;
}


std::vector<int> compatible_cuda_devices() {
    int nDevices;
    checkCuda(hipGetDeviceCount(&nDevices));

    std::vector<int> dev_list;
    hipDeviceProp_t prop;

    for (int i = 0; i < nDevices; i++) {
        checkCuda(hipGetDeviceProperties(&prop, 0));
        if (prop.major >= 6) {
            dev_list.push_back(i);
        }
    }
    if (dev_list.empty()) {
        throw std::runtime_error("No CUDA device with capability minimum 6.0 found");
    }

    return dev_list;
}


double *numpy_to_cuda(py::buffer_info &array_info) {
    double *cuda_ptr;
    double *cpu_ptr = reinterpret_cast<double *>(array_info.ptr);

    size_t bytes = array_info.size * sizeof(double);

    checkCuda(hipMalloc(&cuda_ptr, bytes));
    checkCuda(hipMemcpy(cuda_ptr, cpu_ptr, bytes, hipMemcpyHostToDevice));
    return cuda_ptr;
}


py::array_t<double, py::array::c_style> cuda_to_numpy(py::array::ShapeContainer &shape, double *cuda_ptr) {
    auto numpy_array = py::array_t<double, py::array::c_style>(shape);
    auto array_info = numpy_array.request();

    double *cpu_ptr = reinterpret_cast<double *>(array_info.ptr);

    size_t bytes = array_info.size * sizeof(double);
    checkCuda(hipMemcpy(cpu_ptr, cuda_ptr, bytes, hipMemcpyDeviceToHost));

    return numpy_array;
}

int optimal_threads(int max_threads, int job_size) {
    return MIN(max_threads, job_size);
}

/*
 * Precompute cosine, sine and atheta for each angle.
 * Stored in trigo (cosine, sine, atheta).
 * Grid-stride loop: blocks and threads for angles.
 */
__global__
void precompute_trigo(
        double h,                           /* Sampling step on the image (pixel size) */
        long nI,                            /* Interpolation degree on the Image */
        size_t Nangles,                     /* Number of angles in the sinogram (shape[0]) */
        double s,                           /* Sampling step of the captors (sinogram "pixel size") */
        long nS,                            /* Interpolation degree on the sinogram */
        double *theta,                      /* Projection angles in radian */
        double *trigo                       /* Array containing cosine, sine and atheta for each angle shape {Nangles, 3}*/
) {

    // iterate over the projection angles using blocks and threads alike
    for (long i_angle = blockIdx.x * blockDim.x + threadIdx.x; i_angle < Nangles; i_angle += blockDim.x * gridDim.x) {

        double co = cos(theta[i_angle]);
        double si = sin(theta[i_angle]);

        double atheta = (double) (nI + 1L) / 2.0 * (fabs(si) + fabs(co)) * h + (double) (nS + 1L) / 2.0 * s;

        long index = i_angle * 3;
        trigo[index] = co;
        trigo[index + 1] = si;
        trigo[index + 2] = atheta;
    }
}


/*
 * Precompute the minimum and maximum indexes of sinogram values impacted by each pixel in the image, and the projected coordinates.
 * Stored in sino_bounds (min, max) and t_coord.
 * Grid-stride loop: blocks for angles (Nangles) , threads for image pixels (Nx * Ny).
 */
__global__
void precompute_radon(
        size_t Nx,                          /* Image X size (shape[1]) */
        size_t Ny,                          /* Image Y size (shape[0]) */
        double h,                           /* Sampling step on the image (pixel size) */
        double x0,                          /* Rotation center X in image coordinates */
        double y0,                          /* Rotation center Y in image coordinates */
        size_t Nangles,                     /* Number of angles in the sinogram (shape[0]) */
        size_t Nc,                          /* Number of captors in the sinogram (shape[1]) */
        double s,                           /* Sampling step of the captors (sinogram "pixel size") */
        double t0,                          /* Projection of rotation center */
        double *trigo,                      /* Array containing cosine, sine and atheta for each angle (shape {Nangles, 3}) */
        long *sino_bounds,                  /* Indexes of sinogram impact for all pixels in the image (shape {A, x, y, 2}) */
        double *t_coords                   /* Projected coordinates on the sinogram (shape {A, x, y}) */
) {

    // iterate over the projection angles using blocks
    for (long i_angle = blockIdx.x; i_angle < Nangles; i_angle += gridDim.x) {

        long index_angle = i_angle * 3;

        double co = trigo[index_angle];
        double si = trigo[index_angle + 1];
        double atheta = trigo[index_angle + 2];

        // iterate over the image using threads
        for (long id = threadIdx.x; id < Nx * Ny; id += blockDim.x) {
            long i_x = id / Ny;
            long i_y = id % Ny;

            double x = i_x * h;
            double y = i_y * h;

            // compute the projected coordinate on the sinogram
            double t = t0 + ((x - x0) * co) + ((y - y0) * si);

            // compute the range of sinogram elements impacted by this point and its spline kernel
            long imin = MAX(0L, (long) (ceil((t - atheta) / s)));
            long imax = MIN(Nc - 1L, (long) (floor((t + atheta) / s)));

            // store in the relevant matrices
            auto t_index = i_angle * Nx * Ny + i_x * Ny + i_y;
            t_coords[t_index] = t;

            auto bound_index = t_index * 2;
            sino_bounds[bound_index] = imin;
            sino_bounds[bound_index + 1] = imax;
        }
    }
}

/*
 * Compute the (inverse) radon transform.
 * Grid-stide loop: blocks for angles (Nangles), threads for depth (Nz)
 */
__global__
void cuda_radontransform(
        double *image,                      /* Image (shape {Ny, Nx, Nz})*/
        size_t Nx,
        size_t Ny,
        size_t Nz,
        double *sinogram,                   /* Sinogram (shape (Nangles, Nc, Nz)*/
        size_t Nangles,
        size_t Nc,
        double s,                           /* Sampling step of the captors (sinogram "pixel size") */
        double *kernel,                     /* Kernel table (shape {Nangles, Nt}) */
        size_t Nt,
        double tabfact,                     /* Sampling step of the kernel */
        long *sino_bounds,                  /* Indexes of sinogram impact for all pixels in the image (shape {A, x, y, 2}) */
        double *t_coords,                   /* Projected coordinates on the sinogram (shape {A, x, y}) */
        bool backprojection                 /* Perform a back-projection */
) {

    // iterate over the projection angles
    for (long i_angle = blockIdx.x; i_angle < Nangles; i_angle += gridDim.x) {

        // iterate over the width of the image
        for (long i_x = 0; i_x < Nx; i_x++) {

            // iterate over the height of the image
            for (long i_y = 0; i_y < Ny; i_y++) {

                // fetch the projected coordinate
                auto t_index = i_angle * Nx * Ny + i_x * Ny + i_y;
                auto t = t_coords[t_index];

                // fetch the sinogram bounds
                auto bounds_index = t_index * 2;
                auto imin = sino_bounds[bounds_index];
                auto imax = sino_bounds[bounds_index + 1];

                // iterate over the affected sinogram values
                for (long i_sino = imin; i_sino <= imax; i_sino++) {
                    // compute the position of the point in its spline kernel
                    double xi = fabs((double) i_sino * s - t);
                    long idx = (long) (floor(xi * tabfact + 0.5));

                    for (long i_z = threadIdx.x; i_z < Nz; i_z += blockDim.x) {

                        auto image_index = i_y * Nx * Nz + i_x * Nz + i_z;
                        auto kernel_index = i_angle * Nt + idx;
                        auto sinogram_index = i_angle * Nc * Nz + i_sino * Nz + i_z;

                        if (backprojection) {
                            // update the image
                            atomicAdd(image + image_index, kernel[kernel_index] * sinogram[sinogram_index]);
                        } else {
                            // update the sinogram
                            sinogram[sinogram_index] += (kernel[kernel_index] * image[image_index]);
                        }
                    }
                }
            }
        }
    }
}


py::array_t<double, py::array::c_style> radon_cuda(
        py::array_t<double, py::array::c_style> &image,
        double h,
        long nI,
        double x0,
        double y0,
        py::array_t<double, py::array::c_style> &theta,
        py::array_t<double, py::array::c_style> &kernel,
        double a,
        const long Nc,
        double s,
        long nS,
        double t0
) {

    auto image_info = image.request();
    auto kernel_info = kernel.request();
    auto theta_info = theta.request();

    if (image_info.ndim != 3) {
        throw py::value_error("image must be a 3D array.");
    }

    if (theta_info.ndim != 1) {
        throw py::value_error("theta must be a 1D array.");
    }

    if (kernel_info.ndim != 2) {
        throw py::value_error("kernel must be a 2D array.");
    }

    if (nI < 0L) {
        throw py::value_error("nI must be greater or equal to 0.");
    }

    if (a < 0.0) {
        throw py::value_error("a, the maximal argument of the lookup table must be a positive.");
    }

    if (Nc < 1L) {
        throw py::value_error("The number of captor must at least be 1.");
    }

    if (nS < -1L) {
        throw py::value_error("nS must be greater of equal to -1.");
    }

    const long Nangles = theta_info.shape[0];

    if (Nangles != kernel_info.shape[0]) {
        throw py::value_error("The kernel must have Nangle rows.");
    }

    auto dev_list = compatible_cuda_devices();
    auto device_id = dev_list[0];
    checkCuda(hipSetDevice(device_id));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    auto max_threads = prop.maxThreadsPerBlock;

    double *cuda_theta = numpy_to_cuda(theta_info);

    double *cuda_trigo;
    checkCuda(hipMalloc(&cuda_trigo, Nangles * 3 * sizeof(double)));

    auto n_threads = optimal_threads(max_threads, Nangles);

    precompute_trigo << < 1, n_threads >> > (
            h,
                    nI,
                    Nangles,
                    s,
                    nS,
                    cuda_theta,
                    cuda_trigo
    );

    int Nx = image_info.shape[1];
    int Ny = image_info.shape[0];
    int Nz = image_info.shape[2];

    long *cuda_sino_bounds;
    double *cuda_t_coords;

    n_threads = optimal_threads(max_threads, Nx * Ny);

    checkCuda(hipFree(cuda_theta));

    checkCuda(hipMalloc(&cuda_sino_bounds, Nangles * Nx * Ny * 2 * sizeof(long)));
    checkCuda(hipMalloc(&cuda_t_coords, Nangles * Nx * Ny * sizeof(double)));

    precompute_radon << < Nangles, n_threads >> > (
            Nx,
                    Ny,
                    h,
                    x0,
                    y0,
                    Nangles,
                    Nc,
                    s,
                    t0,
                    cuda_trigo,
                    cuda_sino_bounds,
                    cuda_t_coords
    );

    n_threads = optimal_threads(max_threads, Nz);

    double *cuda_sinogram;
    size_t sinogram_bytes = Nangles * Nc * image_info.shape[2] * sizeof(double);
    auto Nt = kernel_info.shape[1];
    double tabfact = (double) (Nt - 1L) / a;

    checkCuda(hipFree(cuda_trigo));

    // TODO use streams to accelerate data loading
    // TODO combine implementations with radon and iradon (very similar!)
    double *cuda_image = numpy_to_cuda(image_info);
    double *cuda_kernel = numpy_to_cuda(kernel_info);

    checkCuda(hipMalloc(&cuda_sinogram, sinogram_bytes));
    checkCuda(hipMemset(cuda_sinogram, 0, sinogram_bytes));

    cuda_radontransform <<<Nangles, n_threads>>>(
            cuda_image,
                    Nx,
                    Ny,
                    Nz,
                    cuda_sinogram,
                    Nangles,
                    Nc,
                    s,
                    cuda_kernel,
                    kernel_info.shape[1],
                    tabfact,
                    cuda_sino_bounds,
                    cuda_t_coords,
                    false
    );

    checkCuda(hipFree(cuda_image));

    checkCuda(hipFree(cuda_kernel));
    checkCuda(hipFree(cuda_t_coords));
    checkCuda(hipFree(cuda_sino_bounds));

    py::array::ShapeContainer shape = {Nangles, Nc, image_info.shape[2]};

    auto sinogram = cuda_to_numpy(shape, cuda_sinogram);
    checkCuda(hipFree(cuda_sinogram));

    return sinogram;
}

py::array_t<double, py::array::c_style> iradon_cuda(
        py::array_t<double, py::array::c_style> &sinogram,
        double s,
        long nS,
        double t0,
        py::array_t<double, py::array::c_style> &theta,
        py::array_t<double, py::array::c_style> &kernel,
        double a,
        long Nx,
        long Ny,
        double h,
        long nI,
        double x0,
        double y0
) {

    auto sinogram_info = sinogram.request();
    auto kernel_info = kernel.request();
    auto theta_info = theta.request();

    if (sinogram_info.ndim != 3) {
        throw py::value_error("sinogram must be a 3D array.");
    }

    if (theta_info.ndim != 1) {
        throw py::value_error("theta must be a 1D array.");
    }

    if (kernel_info.ndim != 2) {
        throw py::value_error("kernel must be a 2D array.");
    }

    if (nS < 0L) {
        throw py::value_error("nS must be greater or equal to 0.");
    }

    const long Nangles = sinogram_info.shape[0];

    if (Nangles != theta_info.size) {
        throw py::value_error("The number of angles in theta in incompatible with the sinogram.");
    }

    if (nI < -1L) {
        throw py::value_error("nI must be greater or equal to -1.");
    }

    if (Nangles != kernel_info.shape[0]) {
        throw py::value_error("The kernel table must have Nangle rows.");
    }

    if (a < 0) {
        throw py::value_error("a, the max argument of the lookup table must be positive.");
    }

    if (Nx < 1L) {
        throw py::value_error("Nx must at least be 1.");
    }
    if (Ny < 1L) {
        throw py::value_error("Ny must at least be 1.");
    }

    auto dev_list = compatible_cuda_devices();
    auto device_id = dev_list[0];
    checkCuda(hipSetDevice(device_id));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device_id);
    auto max_threads = prop.maxThreadsPerBlock;

    double *cuda_theta = numpy_to_cuda(theta_info);

    double *cuda_trigo;
    checkCuda(hipMalloc(&cuda_trigo, Nangles * 3 * sizeof(double)));

    auto n_threads = optimal_threads(max_threads, Nangles);

    precompute_trigo << < 1, n_threads >> > (
            h,
                    nI,
                    Nangles,
                    s,
                    nS,
                    cuda_theta,
                    cuda_trigo
    );


    int Nz = sinogram_info.shape[2];
    int Nc = sinogram_info.shape[1];

    long *cuda_sino_bounds;
    double *cuda_t_coords;

    n_threads = optimal_threads(max_threads, Nx * Ny);

    checkCuda(hipFree(cuda_theta));

    checkCuda(hipMalloc(&cuda_sino_bounds, Nangles * Nx * Ny * 2 * sizeof(long)));
    checkCuda(hipMalloc(&cuda_t_coords, Nangles * Nx * Ny * sizeof(double)));

    precompute_radon << < Nangles, n_threads >> > (
            Nx,
                    Ny,
                    h,
                    x0,
                    y0,
                    Nangles,
                    Nc,
                    s,
                    t0,
                    cuda_trigo,
                    cuda_sino_bounds,
                    cuda_t_coords
    );

    n_threads = optimal_threads(max_threads, Nz);

    double *cuda_image;
    size_t image_bytes = Ny * Nx * sinogram_info.shape[2] * sizeof(double);
    auto Nt = kernel_info.shape[1];
    double tabfact = (double) (Nt - 1L) / a;

    checkCuda(hipFree(cuda_trigo));

    // TODO use streams to accelerate data loading
    double *cuda_sinogram = numpy_to_cuda(sinogram_info);
    double *cuda_kernel = numpy_to_cuda(kernel_info);

    checkCuda(hipMalloc(&cuda_image, image_bytes));
    checkCuda(hipMemset(cuda_image, 0, image_bytes));

    cuda_radontransform << < Nangles, n_threads >> > (
            cuda_image,
                    Nx,
                    Ny,
                    Nz,
                    cuda_sinogram,
                    Nangles,
                    Nc,
                    s,
                    cuda_kernel,
                    kernel_info.shape[1],
                    tabfact,
                    cuda_sino_bounds,
                    cuda_t_coords,
                    true
    );

    checkCuda(hipFree(cuda_sinogram));

    checkCuda(hipFree(cuda_kernel));
    checkCuda(hipFree(cuda_t_coords));
    checkCuda(hipFree(cuda_sino_bounds));

    py::array::ShapeContainer shape = {Ny, Nx, sinogram_info.shape[2]};

    auto image = cuda_to_numpy(shape, cuda_image);
    checkCuda(hipFree(cuda_image));

    return image;
}
